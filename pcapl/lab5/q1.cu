#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>

__global__ void addVectorsBlockSizeN(int *a, int *b, int *c, int N) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N) {
        c[idx] = a[idx] + b[idx];
    }
}

__global__ void addVectorsNThreads(int *a, int *b, int *c, int N) {
    int idx = threadIdx.x;
    while (idx < N) {
        c[idx] = a[idx] + b[idx];
        idx += blockDim.x;
    }
}

int main() {
    int N = 1024;  // Length of vectors
    int blockSize = N; // Block size equal to N
    int numBlocks = 1; // We use only one block in this case

    int *h_a, *h_b, *h_c; // Host vectors
    int *d_a, *d_b, *d_c; // Device vectors

    // Allocate memory on the host
    h_a = (int*)malloc(N * sizeof(int));
    h_b = (int*)malloc(N * sizeof(int));
    h_c = (int*)malloc(N * sizeof(int));

    // Initialize input vectors on the host
    for (int i = 0; i < N; ++i) {
        h_a[i] = i;
        h_b[i] = i;
    }

    // Allocate memory on the device
    hipMalloc((void**)&d_a, N * sizeof(int));
    hipMalloc((void**)&d_b, N * sizeof(int));
    hipMalloc((void**)&d_c, N * sizeof(int));

    // Copy input vectors from host to device
    hipMemcpy(d_a, h_a, N * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, N * sizeof(int), hipMemcpyHostToDevice);

    // Launch kernel with block size equal to N
    addVectorsBlockSizeN<<<numBlocks, blockSize>>>(d_a, d_b, d_c, N);

    // Copy result vector from device to host
    hipMemcpy(h_c, d_c, N * sizeof(int), hipMemcpyDeviceToHost);

    // Print the first few elements of the result vector
    for (int i = 0; i < 10; ++i) {
        printf("%d + %d = %d\n", h_a[i], h_b[i], h_c[i]);
    }

    // Free memory on the device
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    // Free memory on the host
    free(h_a);
    free(h_b);
    free(h_c);

    return 0;
}
